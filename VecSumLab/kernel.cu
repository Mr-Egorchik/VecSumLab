#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__  
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <random>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>

#define BLOCK_DIM_X 1000

using namespace std;

float get_sum_cpu(float* arr, int len) {
    float sum = 0;
    for (size_t i = 0; i < len; ++i)
        sum += arr[i];
    return sum;
}

__global__ void get_sum_gpu(float* arr, int len, float* res)
{
    __shared__ float temp[BLOCK_DIM_X];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < len) {
        temp[threadIdx.x] = arr[idx];
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        float summ = 0;
        for (int i = 0; i < blockDim.x; ++i)
            summ += temp[i];
        atomicAdd(res, summ);
    }

}

int main()
{
    srand(time(0));

    int len = 1000000;

    float* vec = new float[len];
    for (size_t i = 0; i < len; ++i)
        vec[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

    clock_t start, end;
    float sum_cpu;

    start = clock();
    for (int i = 0; i < 12; ++i)
        sum_cpu = get_sum_cpu(vec, len);
    end = clock();

    double cpu_time = static_cast <double>(end - start) / static_cast <double>(CLOCKS_PER_SEC);

    std::cout << "\nSum on CPU:\t" << sum_cpu << "\nCPU time:\t" << cpu_time / 12;

    float* dvec;
    float* sum_gpu = new float;
    *sum_gpu = 0;
    float* dsum;
    hipMalloc(&dvec, len * sizeof(float));
    hipMalloc(&dsum, sizeof(float));

    hipMemcpy(dvec, vec, len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dsum, sum_gpu, sizeof(float), hipMemcpyHostToDevice);

    dim3 block_dim(BLOCK_DIM_X);
    dim3 grid_dim(ceil(static_cast <float> (len) / static_cast <float> (block_dim.x)));

    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);

    hipEventRecord(begin, 0);
    get_sum_gpu << <grid_dim, block_dim >> > (dvec, len, dsum);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gpu_time;
    hipEventElapsedTime(&gpu_time, begin, stop);

    hipMemcpy(sum_gpu, dsum, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nSum on GPU:\t" << *sum_gpu << "\nGPU time:\t" << gpu_time / 1000.;

    hipFree(dvec);

    
    thrust::device_vector<float> D(len);
    for (size_t i = 0; i < len; ++i)
        D[i] = vec[i];

    hipEventRecord(begin, 0);
    float thrust_sum = thrust::reduce(D.begin(), D.end(), (float)0, thrust::plus<float>());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float thrust_time;
    hipEventElapsedTime(&thrust_time, begin, stop);
    std::cout << "\nSum on thrust:\t" << thrust_sum << "\nThrust time:\t" << thrust_time / 1000.;

    delete[] vec;

    return 0;
}